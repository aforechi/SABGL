#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "vgram.h"
#include "vgram_base.cuh"


__global__ void
cudaNeuronTrain(VG_RAM_WNN vg_ram_wnn, DATA_SET training_set, int sample)
{
	int sample_class;

	sample_class = training_set.d_sample_class[sample];
	for (int neuron = blockIdx.x; neuron < vg_ram_wnn.number_of_neurons; neuron += gridDim.x)
	{
		// Escreve diretamente na memória do neurônio o padrão de bits gerado a partir da sample
		cudaBuildBitPattern((unsigned int *)
				cudaGetNeuronMemoryByNeuronAndSample(
						vg_ram_wnn.d_memories,
						vg_ram_wnn.memory_size,
						vg_ram_wnn.memory_bit_group_size,
						neuron, sample),
						&(vg_ram_wnn.d_synapses[neuron * vg_ram_wnn.number_of_synapses_per_neuron]),
						&(training_set.d_sample[sample * vg_ram_wnn.input_size]),
						vg_ram_wnn.number_of_synapses_per_neuron);

		// Escreve diretamente na memória do neurônio a classe associada ao padrão de bits
		cudaSetNeuronMemory(vg_ram_wnn.d_memories,
				vg_ram_wnn.memory_size,
				vg_ram_wnn.memory_bit_group_size,
				neuron, sample, sample_class);
	}

}


void
Train(VG_RAM_WNN *vg_ram_wnn, DATA_SET *training_set)
{
	int *d_sample;
	int *d_sample_class;
	int *d_synapses;
	int *d_memories;

	cudaCheckError(hipMalloc((int **) &(d_sample_class),
			training_set->num_samples * sizeof(int)));
	cudaCheckError(hipMalloc((int **) &(d_sample),
			training_set->num_inputs * training_set->num_samples * sizeof(int)));

	hipMemcpy(d_sample_class, training_set->sample_class,
			training_set->num_samples * sizeof(int),
			hipMemcpyHostToDevice);
	hipMemcpy(d_sample, training_set->sample,
			training_set->num_samples * training_set->num_inputs * sizeof(int),
			hipMemcpyHostToDevice);

	training_set->d_sample = d_sample;
	training_set->d_sample_class = d_sample_class;

	cudaCheckError(hipMalloc((int **) &(d_synapses),
			vg_ram_wnn->number_of_synapses_per_neuron * vg_ram_wnn->number_of_neurons * sizeof(int)));

	hipMemcpy(d_synapses, vg_ram_wnn->synapses,
			vg_ram_wnn->number_of_neurons * vg_ram_wnn->number_of_synapses_per_neuron * sizeof(int),
			hipMemcpyHostToDevice);

	vg_ram_wnn->d_synapses = d_synapses;

	cudaCheckError(hipMalloc((int**) &(d_memories),
			vg_ram_wnn->number_of_neurons * vg_ram_wnn->memory_size * (vg_ram_wnn->memory_bit_group_size + 1) * sizeof(int)));

	vg_ram_wnn->d_memories = d_memories;

	for (int sample = 0; sample < training_set->num_samples; sample ++)
	{
		cudaNeuronTrain<<<1024,BLOCK_DIM>>>(*vg_ram_wnn,*training_set,sample);
	}
	#ifdef TEST_ON_CPU
	hipMemcpy(vg_ram_wnn->memories, d_memories,
			vg_ram_wnn->number_of_neurons * vg_ram_wnn->memory_size * (vg_ram_wnn->memory_bit_group_size + 1) * sizeof(int),
			hipMemcpyDeviceToHost);
	#endif
	hipFree(d_sample);
	hipFree(d_sample_class);
}


