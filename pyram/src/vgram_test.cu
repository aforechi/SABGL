#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "vgram.h"
#include "vgram_base.cuh"


__device__ int
cudaHammingDistance(unsigned int *bit_pattern1, unsigned int *bit_pattern2,  int memory_bit_group_size)
{
	int i;
	unsigned int bit_difference;
	int hamming_distance;

	hamming_distance = 0;
	for (i = 0; i < memory_bit_group_size; i++)
	{
		bit_difference = bit_pattern1[i] ^ bit_pattern2[i];
		hamming_distance += __popc(bit_difference);
	}
	return hamming_distance;
}


__device__ void
cudaFindNearestPattern(unsigned int *bit_pattern,
		int *neuron_memory,
		int *neuron_output,
		int memory_size,
		int memory_bit_group_size)
{
	int hd_register;
	__shared__ int short_hd;

	int best_hd = 1<<30;
	int best_sample;
	short_hd = 1<<30;

	__syncthreads();

	for (int learned_pattern = threadIdx.x; learned_pattern < memory_size; learned_pattern += blockDim.x)
	{
		hd_register = cudaHammingDistance((unsigned int *)
				cudaGetNeuronMemoryBySample(neuron_memory, memory_bit_group_size, learned_pattern),
				bit_pattern, memory_bit_group_size);

		if(hd_register < best_hd)
		{
			best_hd = hd_register;
			best_sample = learned_pattern;
		}
	}

	atomicMin(&short_hd, best_hd);

	__syncthreads();

	if(short_hd == best_hd)
	{
		*neuron_output = cudaGetNeuronMemoryBySample(neuron_memory, memory_bit_group_size, best_sample)[memory_bit_group_size];
	}

}


__global__ void
cudaNeuronTest(VG_RAM_WNN vg_ram_wnn, DATA_SET testing_set, int sample)
{
	__shared__
	unsigned int bit_pattern[BLOCK_DIM];

	for (int neuron = blockIdx.x; neuron < vg_ram_wnn.number_of_neurons; neuron += gridDim.x)
	{
		cudaBuildBitPattern(bit_pattern,
				&(vg_ram_wnn.d_synapses[neuron * vg_ram_wnn.number_of_synapses_per_neuron]),
				&(testing_set.d_sample[sample * vg_ram_wnn.input_size]),
				vg_ram_wnn.number_of_synapses_per_neuron);

		cudaFindNearestPattern(bit_pattern,
				cudaGetNeuronMemoryByNeuron(vg_ram_wnn.d_memories,
						vg_ram_wnn.memory_size,
						vg_ram_wnn.memory_bit_group_size,
						neuron),
						&(vg_ram_wnn.d_network_output[sample * vg_ram_wnn.number_of_neurons + neuron]),
						vg_ram_wnn.memory_size,
						vg_ram_wnn.memory_bit_group_size);
	}
}


void
Test(VG_RAM_WNN *vg_ram_wnn, DATA_SET *testing_set)
{
	int *d_sample_class;
	int *d_sample;
	int *d_network_output;

	#ifdef TRAIN_ON_CPU
	int *d_synapses;
	int *d_memories;

	cudaCheckError(hipMalloc((int **) &(d_synapses),
			vg_ram_wnn->number_of_synapses_per_neuron * vg_ram_wnn->number_of_neurons * sizeof(int)));

	hipMemcpy(d_synapses, vg_ram_wnn->synapses,
			vg_ram_wnn->number_of_neurons * vg_ram_wnn->number_of_synapses_per_neuron * sizeof(int),
			hipMemcpyHostToDevice);

	vg_ram_wnn->d_synapses = d_synapses;

	cudaCheckError(hipMalloc((int**) &(d_memories),
			vg_ram_wnn->number_of_neurons * vg_ram_wnn->memory_size * (vg_ram_wnn->memory_bit_group_size + 1) * sizeof(int)));

	hipMemcpy(d_memories, vg_ram_wnn->memories,
			vg_ram_wnn->number_of_neurons * vg_ram_wnn->memory_size * (vg_ram_wnn->memory_bit_group_size + 1) * sizeof(int),
			hipMemcpyHostToDevice);

	vg_ram_wnn->d_memories = d_memories;
	#endif

	cudaCheckError(hipMalloc((int **) &(d_sample_class),
			testing_set->num_samples * sizeof(int)));
	cudaCheckError(hipMalloc((int **) &(d_sample),
			testing_set->num_inputs * testing_set->num_samples * sizeof(int)));

	hipMemcpy(d_sample_class, testing_set->sample_class,
			testing_set->num_samples * sizeof(int),
			hipMemcpyHostToDevice);
	hipMemcpy(d_sample, testing_set->sample,
			testing_set->num_samples * testing_set->num_inputs * sizeof(int),
			hipMemcpyHostToDevice);

	testing_set->d_sample = d_sample;
	testing_set->d_sample_class = d_sample_class;

	cudaCheckError(hipMalloc((int **) &(d_network_output),
			vg_ram_wnn->test_size * vg_ram_wnn->number_of_neurons * sizeof(int)));

	vg_ram_wnn->d_network_output = d_network_output;

	for (int sample = 0; sample < testing_set->num_samples; sample++)
	{
		cudaNeuronTest<<<1024,BLOCK_DIM>>>(*vg_ram_wnn,*testing_set,sample);
	}

	hipMemcpy(vg_ram_wnn->network_output, d_network_output,
			vg_ram_wnn->test_size * vg_ram_wnn->number_of_neurons * sizeof(int),
			hipMemcpyDeviceToHost);

	hipFree(d_sample);
	hipFree(d_sample_class);
	hipFree(d_network_output);
	hipFree(vg_ram_wnn->d_synapses);
	hipFree(vg_ram_wnn->d_memories);
}

