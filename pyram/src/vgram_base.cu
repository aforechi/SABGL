#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "vgram.h"
#include "vgram_base.cuh"

#define BLOCK_DIM (PATTERN_UNIT_SIZE*8)

__device__ void
cudaSetNeuronMemory(int *mem, int memory_size, int memory_bit_group_size, int neuron, int sample, int value)
{
	mem[(neuron * memory_size + sample) * (memory_bit_group_size+1) + memory_bit_group_size] = value;
}


__device__ int
cudaGetNeuronMemory(int *mem, int memory_size, int memory_bit_group_size, int neuron, int sample)
{
	return mem[(neuron * memory_size + sample) * (memory_bit_group_size+1) + memory_bit_group_size];
}


__device__ int *
cudaGetNeuronMemoryByNeuronAndSample(int *mem, int memory_size, int memory_bit_group_size, int neuron, int sample)
{
	return &mem[(neuron * memory_size + sample) * (memory_bit_group_size+1)];
}


__device__ int *
cudaGetNeuronMemoryByNeuron(int *mem, int memory_size, int memory_bit_group_size, int neuron)
{
	return &mem[neuron * memory_size * (memory_bit_group_size+1)];
}


__device__ int *
cudaGetNeuronMemoryBySample(int *mem, int memory_bit_group_size, int sample)
{
	return &mem[sample * (memory_bit_group_size+1)];
}


__device__ void
cudaBuildBitPattern(unsigned int *bit_pattern, int *synapses, int *network_input, int number_of_synapses_per_neuron)
{
	int synapse;
	int current_bit_pattern_group;
	int tid = threadIdx.x%PATTERN_UNIT_SIZE;
	__shared__ unsigned int bp[BLOCK_DIM];
	unsigned int aux;

	if(threadIdx.x<BLOCK_DIM) bp[threadIdx.x]=0;

	__syncthreads();

	for (synapse = threadIdx.x; synapse < number_of_synapses_per_neuron -1; synapse += blockDim.x)
	{
		// Cada grupo de padrão de bits tem tamanho igual a PATTERN_UNIT_SIZE.
		current_bit_pattern_group = synapse / PATTERN_UNIT_SIZE;

		// Minchington: Uma sinapse compara com a próxima
		aux = (network_input[synapses[synapse]] > network_input[synapses[synapse+1]]) ? 1 : 0;

		aux  = aux << tid;

		atomicOr(&bp[current_bit_pattern_group],aux);

	}
	__syncthreads();

	if(threadIdx.x == blockDim.x)
	{
		current_bit_pattern_group = synapse / PATTERN_UNIT_SIZE;
		aux  = aux << tid;
		aux = (network_input[synapses[synapse]] > network_input[synapses[0]]) ? 1 : 0;
		atomicOr(&bp[current_bit_pattern_group],aux);
	}

	__syncthreads();

	if(threadIdx.x < number_of_synapses_per_neuron / PATTERN_UNIT_SIZE)
	{
		bit_pattern[threadIdx.x] =  bp[threadIdx.x];
	}
}

void cudaAssert(hipError_t code, const char *file, int line, bool abort)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"Error assert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
